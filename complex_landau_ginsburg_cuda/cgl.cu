#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <complex.h>

// Not using shared memory. 
// Could be made much more efficient by using fft callbacks, 
// but that requires linking to static cuda libraries.
// See: https://devblogs.nvidia.com/cuda-pro-tip-use-cufft-callbacks-custom-data-processing/


// Using a 16x16 grid of gpu threads.
const int num_threads = 16;

// these parameters stay constant throughout after setting. 
__device__ static hipfftDoubleComplex dev_const_c1, dev_const_c3;
__device__ static double dev_const_L;
__device__ static int dev_N;

// Initialize the random kernels
__global__ void init_random(long int seed, hiprandState_t *states){
    int idx = threadIdx.x + blockIdx.x * num_threads;
    int idy = threadIdx.y + blockIdx.y * num_threads; 
    int id = idy * dev_N + idx;
    if (idx < dev_N && idy < dev_N){
        hiprand_init(seed, id, 0, &states[id]);
    }
}

// Generate the random values for the array A. 
// Also precalculate the x and y factors that we use in the derivative calculation.
__global__ void init_threads(hiprandState_t *states, hipfftDoubleComplex* A,
                            double* xfactors, double* yfactors){
    int idx = threadIdx.x + blockIdx.x * num_threads;
    int idy = threadIdx.y + blockIdx.y * num_threads; 
    int id = idy * dev_N + idx;
    if (idx < dev_N && idy < dev_N){
        double real = 3 * hiprand_uniform(&states[id]) - 1.5;
        double imag = 3 * hiprand_uniform(&states[id]) - 1.5;
        A[id].x = real;
        A[id].y = imag;
        // precalculating the x and y factors.
        xfactors[id] = pow(fmin(idx, dev_N/1.0 - idx),2);
        yfactors[id] = pow(fmin(idy, dev_N/1.0 - idy),2);
    }
}

// To calculate the derivative (del2A), we need to 
// 1) perform fft
// 2) do some manipulation
// 3) perform ifft
// The fft and ifft are handled separately. 
// This function does only the in-between manipulation.
__global__ void del2A_between_fft(hipfftDoubleComplex* out, double* xfactors, 
                                double* yfactors, hipfftDoubleComplex* in){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y; 
    int id = idy * dev_N + idx;
    if (idx < dev_N && idy < dev_N){
        double temp1 = -(xfactors[id] + yfactors[id]) / (dev_N * dev_N);
        out[id].x = temp1 * in[id].x;
        out[id].y = temp1 * in[id].y;
    }
}

// Final step in each rk4 iteration. 
// The paramter "a_iter" stays constant throughout the 4 rk4 steps, and is updated only at final step.
// "a_step" changes in between the steps
__global__ void rk4_step_finalize(hipfftDoubleComplex *a_iter, hipfftDoubleComplex* a_step,
                                    hipfftDoubleComplex* a_out, double mult_const,
                                    hipfftDoubleComplex* del2a){
    
    int idx = threadIdx.x + blockIdx.x * num_threads;
    int idy = threadIdx.y + blockIdx.y * num_threads; 
    int id = idy * dev_N + idx;
    if (idx < dev_N && idy < dev_N){
        hipfftDoubleComplex term1, term2, term3, term4;
        double absval;

        term1 = a_step[id];
        absval = hipCabs(term1);
        term2 = hipCmul(dev_const_c1, del2a[id]);
        term2.x = term2.x * dev_const_L;
        term2.y = term2.y * dev_const_L;
        
        term3 = hipCmul(term1, dev_const_c3);
        term3.x = -term3.x * absval * absval;
        term3.y = -term3.y * absval * absval;

        term4 = hipCadd(term1, hipCadd(term2, term3));
        term4.x = term4.x * mult_const;
        term4.y = term4.y * mult_const;
        a_out[id] = hipCadd(a_iter[id], term4);
    }
}



int main(int argc, char* argv[]){
    
    // Required command line inputs
    int N = 128; // Number of grid points per dimension
    double c1 = 1.5; // equation coefficient c1
    double c3 = 0.25; // equation coefficient c3
    int iterations = 100000; // Total number of iterations (dt * T) - should be much larger than T

    // Fixed parameters
    int T = 10000; // Number of time units
    double dt = (double)T / iterations; // timestep per iteration
    double L = 128. * M_PI; // length of domain on each side
    
    
    const char* fname = "CGL.out";
    
    if (argc > 1){
        N = (int)atoi(argv[1]);
    }
    if (argc > 2){
        c1 = (double)atof(argv[2]);
    }
    if (argc > 3){
        c3 = (double)atof(argv[3]);
    }
    if (argc > 4){
        iterations = (int)atoi(argv[4]);
    }
    if (argc > 5){
        T = (int)atol(argv[5]);
    }
    if (argc == 6){
        fname = argv[6];
    }
#ifdef VERBOSE
    printf("Parameters are:\n");
    printf("Number of grid points: %d\n", N);
    printf("Coefficient c1: %f\n",c1);
    printf("Coefficient c3: %f\n", c3);
    printf("Number of iterations (M): %d\n", iterations);
    printf("Total system timesteps (T) : %d\n", T);
    printf("Output file name: %s\n", fname);
#endif
    
    FILE* output = fopen(fname, "w"); // Output file

    clock_t begin = clock();
    
    // Using an even grid 
    int num_blocks = N / num_threads + (N % num_threads ? 1 : 0);

    // Calculating some constant parameters for later use.
    hipfftDoubleComplex const_c1;
    const_c1.x = 1;
    const_c1.y = c1;
    hipfftDoubleComplex const_c3;
    const_c3.x = 1;
    const_c3.y = -c3;
    double const_L = (2. * M_PI / L) * (2 * M_PI / L);
    
    // Setting up the constants in the gpu memory.
    hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_const_c1), &const_c1, sizeof(hipfftDoubleComplex));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_const_c3), &const_c3, sizeof(hipfftDoubleComplex));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_const_L), &const_L, sizeof(double));
    
    // setting up the mesh
    dim3 meshBlocks(num_blocks, num_blocks);
    dim3 meshThreads(num_threads, num_threads);
    
    // initializing the random kernels in each thread
    hiprandState_t* dev_states;
    hipMalloc((void**)&dev_states, N * N * sizeof(hiprandState));
    init_random<<<meshBlocks, meshThreads>>>(time(NULL), dev_states);
    
    // this is the actual grid array
    hipfftDoubleComplex* A = (hipfftDoubleComplex*)malloc(N * N * sizeof(hipfftDoubleComplex));
    
    // more setup for gpu
    hipfftDoubleComplex* dev_A;
    double* dev_xfactors;
    double* dev_yfactors;
    hipMalloc((void**)&dev_A, N * N * sizeof(hipfftDoubleComplex));
    hipMalloc((void**)&dev_xfactors, N * N * sizeof(double));
    hipMalloc((void**)&dev_yfactors, N * N * sizeof(double));

    // get the initial random values for A, and also calculate the x and y factors for later use in del2A calc
    init_threads<<<meshBlocks, meshThreads>>>(dev_states, dev_A, dev_xfactors, dev_yfactors);

    // setting up the fft
    hipfftHandle plan;
    hipfftPlan2d(&plan, N, N, HIPFFT_Z2Z);
    
    // creating some temporary arrays for shifting data in between rk4 steps
    hipfftDoubleComplex* dev_temp_a1;
    hipfftDoubleComplex* dev_temp_a2;
    hipMalloc((void**)&dev_temp_a1, N * N * sizeof(hipfftDoubleComplex));
    hipMalloc((void**)&dev_temp_a2, N * N * sizeof(hipfftDoubleComplex));
    
    hipfftDoubleComplex* dev_a_step;
    hipMalloc((void**)&dev_a_step, N * N * sizeof(hipfftDoubleComplex));
    
    
    hipMemcpy(A, dev_A, N * N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    fwrite(A, sizeof(hipfftDoubleComplex), N*N, output);
    
    for (int i=0; i< 10; i++){
        for (int j=0; j < iterations / 10; ++j){

            // rk4 step 1
            hipfftExecZ2Z(plan, dev_A, dev_temp_a1, HIPFFT_FORWARD);
            del2A_between_fft<<<meshBlocks, meshThreads>>>(dev_temp_a2, dev_xfactors, dev_yfactors, dev_temp_a1);
            hipfftExecZ2Z(plan, dev_temp_a2, dev_temp_a1, HIPFFT_BACKWARD);
            rk4_step_finalize<<<meshBlocks, meshThreads>>>(dev_A, dev_A, dev_a_step, dt / 4., dev_temp_a1);

            // step 2
            hipfftExecZ2Z(plan, dev_a_step, dev_temp_a1, HIPFFT_FORWARD);
            del2A_between_fft<<<meshBlocks, meshThreads>>>(dev_temp_a2, dev_xfactors, dev_yfactors, dev_temp_a1);
            hipfftExecZ2Z(plan, dev_temp_a2, dev_temp_a1, HIPFFT_BACKWARD);
            rk4_step_finalize<<<meshBlocks, meshThreads>>>(dev_A, dev_a_step, dev_a_step, dt/3, dev_temp_a1);

            // step 3
            hipfftExecZ2Z(plan, dev_a_step, dev_temp_a1, HIPFFT_FORWARD);
            del2A_between_fft<<<meshBlocks, meshThreads>>>(dev_temp_a2, dev_xfactors, dev_yfactors, dev_temp_a1);
            hipfftExecZ2Z(plan, dev_temp_a2, dev_temp_a1, HIPFFT_BACKWARD);
            rk4_step_finalize<<<meshBlocks, meshThreads>>>(dev_A, dev_a_step, dev_a_step, dt/2, dev_temp_a1);


            // step 4
            hipfftExecZ2Z(plan, dev_a_step, dev_temp_a1, HIPFFT_FORWARD);
            del2A_between_fft<<<meshBlocks, meshThreads>>>(dev_temp_a2, dev_xfactors, dev_yfactors, dev_temp_a1);
            hipfftExecZ2Z(plan, dev_temp_a2, dev_temp_a1, HIPFFT_BACKWARD);
            rk4_step_finalize<<<meshBlocks, meshThreads>>>(dev_A, dev_a_step, dev_A, dt, dev_temp_a1);
        }
        hipMemcpy(A, dev_A, N * N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
        fwrite(A, sizeof(hipfftDoubleComplex), N*N, output);
    }
    printf("\n\nruntime %g\n\n", (double)(clock()-begin)/CLOCKS_PER_SEC);
    fclose(output);
    
    hipfftDestroy(plan);
    hipFree(dev_temp_a1);
    hipFree(dev_temp_a2);
    hipFree(dev_xfactors);
    hipFree(dev_yfactors);
    hipFree(dev_a_step);
    hipFree(dev_A);

    free(A);
    return 0;
}