#include "hip/hip_runtime.h"
#include "funcs.h"

double linspace(double initval,
	      double finalval,
	      int N,
	      double* gridpoints)
{
  double diff = (finalval - initval) / (N-1);

  gridpoints[0] = initval;
  for (int i=1; i<N; i++)
    gridpoints[i] = gridpoints[i-1] + diff;

  return diff;
}
/*
void gvalsCalc(double* xpoints,
	       double* ypoints,
	       int nx,
	       int ny,
	       double lambda,
	       double* gvals){
  double mfactor = 10. * lambda / sqrt(M_PI);
  double l2 = lambda * lambda;
  int indx;
  double xhigh, xhighsq, xlow, xlowsq, term1, term2, ysq;
  
  for (int i=0; i<ny; i++){
    for (int j=0; j<nx; j++){
      indx = i * nx + j;
      xlow = xpoints[j] - 1;
      xlowsq = xlow * xlow;
      
      xhigh = xpoints[j] + 1;
      xhighsq = xhigh * xhigh;

      ysq = ypoints[i] * ypoints[i];
      
      term1 = exp(-l2 * (xlowsq + ysq));
      term2 = exp(-l2 * (xhighsq + ysq));

      gvals[indx] = mfactor * term1 - mfactor * term2;
    }
  }
}
*/

void gvalsCalc(double x_min, double x_max, double y_min, double y_max, 
                int xpoints, int ypoints, double lambda, double* gvals, 
                double* dx, double* dy){

    *dx = (x_max - x_min) / (xpoints - 1);
    *dy = (y_max - y_min) / (ypoints - 1);
    double mfactor = 10. * lambda / sqrt(M_PI);
    double xval, xlow, xlowsq, xhigh, xhighsq, yval, ysq, term1, term2;
    
    for (int i=0; i < ypoints; i++){
      for (int j=0; j<xpoints; j++){

        xval = x_min + j * (*dx);
        xlow = xval - 1;
        xlowsq = xlow * xlow;
        xhigh = xval + 1;
        xhighsq = xhigh * xhigh;

        yval = y_min + i *  (*dy);
        ysq = yval * yval;

        term1 = exp(-lambda * lambda * (xlowsq + ysq));
        term2 = exp(-lambda * lambda * (xhighsq + ysq));

        gvals[i * xpoints + j] = mfactor * term1 - mfactor * term2;
      }
    }
}

void init2dArray(int nx, int ny, double* grid)
{
  for (int i=0; i<nx; i++)
    {
      for (int j=0; j<ny; j++){
	grid[i * ny + j] = 0;
      }
    }
}	      

void printArray(int N, double* array)
{
  for (int i=0; i<N; i++)
    printf("%d %f\n", i, array[i]);
}


void print2dArray(int nx, int ny, double* grid)
{
  for (int i=0; i<ny; i++)
    {
      for (int j=0; j<nx; j++)
	printf("%f  ",grid[nx *i + j]);
      printf("\n");
    }
}


