#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <ctype.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "funcs.h"
#include "handleerror.h"

// Using the RED-BLACK alternating update scheme. 
#define RED 0
#define BLACK 1

// Using a 2d grid of 16x16 threads. This grid should be safe in all modern gpus.
// Each thread corresponds to a point in the NxM grid.
const int num_threads = 16;

// declare constant memory on the device. these parameters stay constant throughout.
__device__ static int dev_N, dev_M;
__device__ static double dev_omega, dev_lambda, dev_dx, dev_dy;

// iterate is the gpu kernel that handles the gpu calls.
// the "red" parameter tells the function to alternate between red and black points in the grid.
__global__ void iterate(double* gvals, double *u, double* maxresid_per_block, int red);

// update is a gpu-only function that calculates the new "u" value
// and the residue per thread
__device__ void update(double* localu, double* localu_new, double* localg, 
                        double* resid_per_thread);

// initialize is a gpu-only function that copies the u and g values from the global memory
// to the local memory.
__device__ void initialize(double* localu, double* localu_new, double* localg, 
                            double* gvals, double* u);

// takes the residual obtained per thread and calculates the maximum value for each block
__device__ double getMaxResidPerBlock(double* resid_per_thread);
 
int main(int argc, char* argv[]){

    /* Setting up from the input parameters */
    //=====================================================================================
    int N = 128; // Number of y points
    double omega = 1.5; // Value of relaxation parameter.
    double tol = 1e-9;  // Tolerance. Default value is 1e-9.
    int maxiter = 1000; // Maximum number of iterations.
    const char* fname = "sources.out"; // Output file name
    double lambda = 100.; // lambda parameter.

    if (argc > 1)
        N = atoi(argv[1]);
    if (argc > 2)
        omega = atof(argv[2]);
    if (argc > 3){
        tol = atof(argv[3]);
    }
    if (argc > 4){
        maxiter = atoi(argv[4]);
    }
    if (argc > 5){
        fname = argv[5];
    }
    if (argc > 6){
        lambda = atof(argv[6]);
    }

    int M = 2 * N - 1;

#ifdef VERBOSE
    // Using a preprocessor flag to control verbose/debug output
    printf("Parameters are:\n");
    printf("number of y-points (N) : %d\n", N);
    printf("number of x-points (M) : %d\n", M);
    printf("omega : %f\n", omega);
    printf("tolerance: %f\n", tol);
    printf("max iterations: %d\n", maxiter);
    printf("Output fname: %s\n", fname);
    printf("lambda: %f\n", lambda);
#endif

double *gvals = (double*)malloc(M * N * sizeof(double));
double* u = (double*)malloc(N * M * sizeof(double));
double dx, dy, maxresid;

double x_min = -2., x_max = 2., y_min=-1, y_max=1;

gvalsCalc(x_min, x_max, y_min, y_max, M, N, lambda, gvals, &dx, &dy);
init2dArray(M, N, u);
maxresid = 1 + tol;

    // completed initial parameter setup.
    //========================================================================================
    clock_t begin = clock();

    // Setting the number of blocks. The blocks at the edges can have threads that are not utilized

    int num_blocks_x = M / num_threads + (M % num_threads ? 1 : 0);
    int num_blocks_y = N / num_threads + (N % num_threads ? 1 : 0);
#ifdef VERBOSE
    printf("threads %d\n", num_threads);
    printf("blocks %d %d\n", num_blocks_x, num_blocks_y);
#endif 

    
    double* resid_per_block_red = (double*)malloc(num_blocks_x * num_blocks_y * sizeof(double));
    double* resid_per_block_black = (double*)malloc(num_blocks_x * num_blocks_y * sizeof(double));

    // Copy the constants to the constant memory on the gpu
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_M), &M, sizeof(int)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_omega), &omega, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_lambda), &lambda, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &dx, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_dy), &dy, sizeof(double)));

    // initialize the global memory on the gpu.
    double* dev_u;
    double* dev_gvals;
    double* dev_resid_per_block; // the max residual is calculated up to per block within the gpu 

    HANDLE_ERROR(hipMalloc((void**)&dev_u, N * M * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_gvals, N * M * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_resid_per_block, M * N * sizeof(double)));

    HANDLE_ERROR(hipMemcpy(dev_u, u, M * N * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_gvals, gvals, M * N * sizeof(double), 
                            hipMemcpyHostToDevice));
    
    // Setting up the blocks/threads grid.
    dim3 meshBlocks(num_blocks_x, num_blocks_y);
    dim3 meshThreads(num_threads, num_threads);
    
    int iter;
    for (iter=0; (iter < maxiter && maxresid > tol); iter++){
        
        // The actual update iterations. The RED and BLACK points are updated with sequential kernel calls.
        // Separately storing the residual per block from the red and black points. 
        iterate<<<meshBlocks, meshThreads>>>(dev_gvals, dev_u, dev_resid_per_block, RED);
        HANDLE_ERROR(hipMemcpy(resid_per_block_red, dev_resid_per_block, num_blocks_x * num_blocks_y * sizeof(double), 
                                hipMemcpyDeviceToHost));

        iterate<<<meshBlocks, meshThreads>>>(dev_gvals, dev_u, dev_resid_per_block, BLACK);
        HANDLE_ERROR(hipMemcpy(resid_per_block_black, dev_resid_per_block, num_blocks_x * num_blocks_y * sizeof(double), 
                                hipMemcpyDeviceToHost));
        
        // Comparing all residuals from the red and black points to calculate the maximum residual.
        maxresid = 0;
        for(int by=0; by<num_blocks_y; ++by){
            for(int bx=0; bx<num_blocks_x; ++bx){
                maxresid = fmax(resid_per_block_red[by * num_blocks_x + bx], maxresid);
                maxresid = fmax(resid_per_block_black[by * num_blocks_x + bx], maxresid);
            }
        }
#ifdef VERBOSE
        printf("iter %d maxresid %f\n", iter, maxresid);
#endif
    }
    // Copying back the final grid points
    HANDLE_ERROR(hipMemcpy(u, dev_u, M * N * sizeof(double), 
                hipMemcpyDeviceToHost));
   
    printf("runtime %g\n", (double)(clock()-begin)/CLOCKS_PER_SEC);

    FILE* output = fopen(fname, "w");
    fwrite(u, sizeof(double), N * M, output);
    fclose(output);

    free(u);
    free(gvals);
    free(resid_per_block_red);
    free(resid_per_block_black);
    HANDLE_ERROR(hipFree(dev_u));
    HANDLE_ERROR(hipFree(dev_gvals));
    HANDLE_ERROR(hipFree(dev_resid_per_block));
    return 0;
}

// This is the workhorse function..
__global__ void iterate(double* gvals, double* u, double* resid_per_block, int red){
    
    // using the shared memory for information needed blockwise.
    __shared__ double localu[(num_threads + 2) * (num_threads + 2)];
    __shared__ double localg[num_threads * num_threads];

    // this is where the updated gridpoints are stored, before copying back to the global memory.
    __shared__ double localu_new[num_threads * num_threads];

    // initializing the array of residuals
    // The blocks at the edges can have unutilized threads, which don't update the corresponding
    // item in the array of residuals. 
    // To avoid using uninitialized values for the residuals, set them all to zero in advance.
    __shared__ double resid_per_thread[num_threads * num_threads];
    resid_per_thread[threadIdx.y * num_threads + threadIdx.x] = 0;

    int g_ix = blockIdx.x * num_threads + threadIdx.x;
    int g_iy = blockIdx.y * num_threads + threadIdx.y;
    int g_i = g_iy * dev_M + g_ix;
    // Ensuring that the calculation only happens at the threads that correspond to actual grid points.
    if (g_ix < dev_M && g_iy < dev_N){
        int t_i = threadIdx.y * num_threads + threadIdx.x; 
        
        // copying data from global to shared memory
        initialize(localu, localu_new, localg, gvals, u);
        __syncthreads();

        // This is an XOR trick to select only red or only black values per iteration.
        int cond2 = ((g_ix % 2 == 0) && (g_iy % 2 == 0));
        int cond3 = ((g_ix % 2 != 0) && (g_iy % 2 != 0));        
        if (red != (cond2 || cond3) ){
            // the actual updates
            update(localu, localu_new, localg, resid_per_thread);
        }

        // copy back to global memory.
        u[g_i] = localu_new[t_i];
        
        __syncthreads();

        // calculate the maximum residue per block from the individual threads.
        if (threadIdx.x == 0 && threadIdx.y == 0){
            resid_per_block[blockIdx.y * gridDim.y + blockIdx.x] = getMaxResidPerBlock(resid_per_thread);
        }
    }
    __syncthreads();
}

__device__ void update(double* localu, double* localu_new, double* localg, double* resid_per_thread){
    int l_ix = threadIdx.x + 1;
    int l_iy = threadIdx.y + 1;
    int l_dim = num_threads + 2;
    int l_i = l_iy * l_dim + l_ix;
    int t_i = threadIdx.y * num_threads + threadIdx.x;
    int g_ix = blockIdx.x * num_threads + threadIdx.x;
    int g_iy = blockIdx.y * num_threads + threadIdx.y;

    double usum = 0;

    // The conditionals are to make sure that we are only using values from within the grid (not periodic)
    // for the updates.
    if (g_iy > 0){ // for y =0, we update from points indexed as y-1
        usum += localu[(l_iy - 1) * l_dim + l_ix] - localu[l_i];
    }

    if (g_iy < (dev_N - 1)){ // for y = (N -1)
        usum += localu[(l_iy + 1) * l_dim + l_ix] - localu[l_i];
    }

    if (g_ix > 0){ // for x=0
        usum += localu[l_iy * l_dim + l_ix -1] - localu[l_i];
    }
    if (g_ix < (dev_M - 1)){ // for x = (M - 1)
        usum += localu[l_iy * l_dim + l_ix + 1] - localu[l_i];
    }
    
    double resid = 0.25 * (usum - dev_dx * dev_dy * localg[t_i]);
    localu_new[t_i] = localu[l_i] + dev_omega * resid;
    resid_per_thread[t_i] = resid;
}

__device__ void initialize(double* localu, double* localu_new, double* localg, double* gvals, double* u){
    int g_ix = blockIdx.x * num_threads + threadIdx.x;
    int g_iy = blockIdx.y * num_threads + threadIdx.y;
    int g_i = g_iy * dev_M + g_ix;
    
    // each block contains an additional two rows and two columns of ghost points at the edges.
    // the actual update points are indexed by l_ix.
    int l_ix = threadIdx.x + 1;
    int l_iy = threadIdx.y + 1;
    int l_dim = num_threads + 2;
    int l_i = l_iy * l_dim + l_ix;

    // For convenience, we assume that the grid is periodic for the initialization only.
    // the updates don't assume the same.
    int g_imx = (g_ix + dev_M -1) % dev_M;
    int g_imy = (g_iy + dev_N -1) % dev_N;
    int g_ipx = (g_ix + 1) % dev_M;
    int g_ipy = (g_iy + 1) % dev_N;
    int t_i = threadIdx.y * num_threads + threadIdx.x; 

    localu[l_i] = u[g_i];
    localu_new[t_i] = u[g_i];
    localg[t_i] = gvals[g_i];
    // The threads at the edges additionally initialize the adjacent ghost point.
    if (threadIdx.x == 0){
        localu[l_iy * l_dim] = u[g_iy * dev_M + g_imx];
    }
    if (threadIdx.x == num_threads - 1){
        localu[l_iy * l_dim + l_ix + 1] = u[g_iy * dev_M + g_ipx];
    }
    if (threadIdx.y == 0){
        localu[l_ix] = u[g_imy * dev_M + g_ix];
    }
    if (threadIdx.y == num_threads - 1){
        localu[(l_iy + 1) * l_dim + l_ix] = u[g_ipy * dev_M + g_ix];
    }
}

__device__ double getMaxResidPerBlock(double* resid_per_thread){
    double maxresid = 0;
        for (int ry=0; ry < num_threads; ry++){
            for (int rx=0; rx < num_threads; rx++){
                maxresid = fmax(maxresid, fabs(resid_per_thread[ry * num_threads + rx]));
            }
        }
    return maxresid;
}